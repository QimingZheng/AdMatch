#include "hip/hip_runtime.h"
#include "src/nfa_kernels.h"

__constant__ int c_transition_offset[SYMBOL_COUNT+1];

// iNFAnt traversal algorithm to process multiple strings on a NFA
// input                        :  total input string 
// input_offset                 :  offset of each input string
// transition_list              :  list of (source, destination) tuples
// transition_offset            :  index of first transition trigger by each symbol
// init_states_vector           :  vector of initial states 
// persis_states_vector         :  vector of persistent states
// final_states_vector          :  vector of final states
// vector_len                   :  length of state vector (# of ST_BLOCKs)   
__global__ void TKO_kernel(unsigned char *input,
                           int *input_offset,
                           Transition *transition_list,
   //                        int *transition_offset,
                           ST_BLOCK *init_states_vector,
                           ST_BLOCK *final_states_vector,
                           int *top_k_offset_per_symbol,
                           ST_BLOCK *lim_vector,
                           int vector_len)
{       
        // Skip to the right input string 
        input += input_offset[block_ID];
        // Get the size of current input string
        int input_bytes = input_offset[block_ID + 1] - input_offset[block_ID];

        extern __shared__ ST_BLOCK s_data[];    // shared memory 

        ST_BLOCK *current_st_vec = s_data;                              // current active states in shared memory
        ST_BLOCK *future_st_vec = s_data + vector_len;                  // future active states in shared memory
        ST_BLOCK *workspace_vec = s_data + 2 * vector_len;              // workspace states in shared memory, helper
        //int *s_transition_offset = (int*)(s_data + 3 * vector_len);     // transition offset in shared memory

        Transition tuple = transition_list[0];                
        ST_T src_state, dst_state;      
        ST_BLOCK src_bit, dst_bit;      
        unsigned int src_block, dst_block;
        int c, transition_start, transition_count, wb_transition_start, wb_transition_count;

        // Copy initial and persistent states from global memory into shared memory
        for (int i = thread_ID; i < vector_len; i += thread_count) {
                current_st_vec[i] = init_states_vector[i];
                workspace_vec[i] = 0;
        }

        __syncthreads();

        // First transition and # of transitions triggered by word boundary
        wb_transition_start = c_transition_offset[WORD_BOUNDARY];
        wb_transition_count = c_transition_offset[WORD_BOUNDARY + 1] - wb_transition_start;
        
        if (wb_transition_count == 0)
                goto BYPASS_HEAD;

        // If the first character is a word character, there is a word boundary before the first character
        if (!is_word_char(input[0]))
                goto BYPASS_HEAD;
                
        // For each transition triggered by word boundary 
        for (int i = thread_ID; i < wb_transition_count; i += thread_count) {
                tuple = transition_list[i + wb_transition_start];
                src_state = tuple.src;
                dst_state = tuple.dst;
                src_bit = 1 << (src_state % bit_sizeof(ST_BLOCK));      // index of state bit inside the block
                dst_bit = 1 << (dst_state % bit_sizeof(ST_BLOCK));
                src_block = src_state / bit_sizeof(ST_BLOCK);           // index of state block
                dst_block = dst_state / bit_sizeof(ST_BLOCK);
                
                // If transition source is set in current active state vector (divergence happens here)
                if (src_bit & current_st_vec[src_block]) {
                        // Set transition destination in CURRENT active state vector
                        atomicOr(&current_st_vec[dst_block], dst_bit);
                }
        }

        __syncthreads();

BYPASS_HEAD:
        // For each byte in the input string
        for (int byt = 0; byt < input_bytes; byt++) {
                // clean future state vector
                for (int i = thread_ID; i < vector_len; i += thread_count) {
                        future_st_vec[i] = 0;
                }
                __syncthreads();

                c = (int)(input[byt]);

                for(int i = 0; i<TOP_K; i++)
                {
                        int offset = top_k_offset_per_symbol[c*TOP_K+i];
                        for(int j=thread_ID; j<vector_len; j+=thread_count){
                                workspace_vec[j] = lim_vector[c*vector_len*TOP_K+i*vector_len+j] & current_st_vec[j];
                        }
                        __syncthreads();
                        
                        int sign = 1-2*(offset<0); // -1-> negative 1->positive
                        int left_1 = max(int(0), int(offset/bit_sizeof(ST_BLOCK)));
                        int right_1 = min(int(vector_len - 1), int(vector_len - 1 + (offset/bit_sizeof(ST_BLOCK))));
                        int left_2 = max(int(0), int(offset/bit_sizeof(ST_BLOCK) + sign));
                        int right_2 = min(int(vector_len-1), int(vector_len- 1 + sign + (offset/bit_sizeof(ST_BLOCK))));

                        if (offset>=0){
                                for(int j = left_1 + thread_ID; j<=right_1; j+=thread_count){
                                        future_st_vec[j] |= 
                                        (workspace_vec[j-offset/bit_sizeof(ST_BLOCK)]<<(offset%bit_sizeof(ST_BLOCK)));
                                }
                                __syncthreads();

                                for(int j = left_2 + thread_ID; j <= right_2; j+=thread_count){
                                        future_st_vec[j] |= 
                                        (workspace_vec[j-offset/bit_sizeof(ST_BLOCK)-1]>>(bit_sizeof(ST_BLOCK)-(offset%bit_sizeof(ST_BLOCK))));
                                }
                                __syncthreads();

                        }
                         else{
                                for(int j = left_1 + thread_ID; j<=right_1; j+=thread_count){
                                        future_st_vec[j] |= 
                                        (workspace_vec[j-(offset/bit_sizeof(ST_BLOCK))]>>(((-offset)%bit_sizeof(ST_BLOCK))));
                                }
                                __syncthreads();

                                for(int j = left_2 + thread_ID; j <= right_2; j+=thread_count){
                                        future_st_vec[j] |= 
                                        (workspace_vec[j-(offset/bit_sizeof(ST_BLOCK))+1]<<((bit_sizeof(ST_BLOCK)-((-offset)%bit_sizeof(ST_BLOCK)))));
                                }
                                __syncthreads();                                
                        }

                }

                transition_start = c_transition_offset[c];
                transition_count = c_transition_offset[c + 1] - transition_start;

                // For each transition triggered by the character
                for (int i = thread_ID; i < transition_count; i += thread_count) {
                        tuple = transition_list[i + transition_start];
                        src_state = tuple.src;
                        dst_state = tuple.dst;
                        src_bit = 1 << (src_state % bit_sizeof(ST_BLOCK));      // index of state bit inside the block
                        dst_bit = 1 << (dst_state % bit_sizeof(ST_BLOCK));
                        src_block = src_state / bit_sizeof(ST_BLOCK);           // index of state block
                        dst_block = dst_state / bit_sizeof(ST_BLOCK);

                        // If transition source is set in current active state vector (divergence happens here)
                        if (src_bit & current_st_vec[src_block]) {
                                // Set transition destination in future active state vector
                                atomicOr(&future_st_vec[dst_block], dst_bit);
                        }
                }

                // Swap current and future active state vector
                if (current_st_vec == s_data) {
                        current_st_vec = s_data + vector_len;
                        future_st_vec = s_data;
                } else {
                        current_st_vec = s_data;
                        future_st_vec = s_data + vector_len;
                }

                __syncthreads();

                // No transition triggered by word boundary
                if (wb_transition_count == 0)
                        continue;

                // If there is NOT a word boundary between input[byt] and input[byt + 1] or after the last character
                if ((byt < input_bytes - 1 && (is_word_char(input[byt]) ^ is_word_char(input[byt + 1])) == 0) ||
                    (byt == input_bytes - 1 && !is_word_char(input[input_bytes - 1])))
                        continue;

                // For each transition triggered by word boundary
                for (int i = thread_ID; i < wb_transition_count; i += thread_count) {
                        tuple = transition_list[i + wb_transition_start];
                        src_state = tuple.src;   
                        dst_state = tuple.dst;
                        src_bit = 1 << (src_state % bit_sizeof(ST_BLOCK));      // index of state bit inside the block
                        dst_bit = 1 << (dst_state % bit_sizeof(ST_BLOCK));
                        src_block = src_state / bit_sizeof(ST_BLOCK);           // index of state block
                        dst_block = dst_state / bit_sizeof(ST_BLOCK);
                
                        // If transition source is set in current active state vector (divergence happens here)
                        if (src_bit & current_st_vec[src_block]) {
                                // Set transition destination in CURRENT active state vector
                                atomicOr(&current_st_vec[dst_block], dst_bit);
                        }
                }

                __syncthreads();
        }

        // Copy final active states from shared memory into global memory
        for (int i = thread_ID; i < vector_len; i += thread_count) {
                final_states_vector[block_ID * vector_len + i] = current_st_vec[i];
        }        
}

// Host function to run iNFAnt algorithm on GPU
// This function can process multiple strings on a NFA simultaneously
// tg                   :  NFA transition graph
// h_input_array        :  array of input string in host memory
// input_bytes_array    :  array of string length
// array_size           :  array size (# of strings to match)
// threads_per_block    :  # of threads per block for kernel function 
// show_match_result    :  print regex matching result if this variable is true                     
vector<int>* run_TKO(class TransitionGraph *tg, 
             unsigned char **h_input_array, 
             int *input_bytes_array, 
             int array_size,
             int threads_per_block, 
             bool show_match_result,
             bool profiler_mode)
{
        struct timeval start_time, end_time;
        hipEvent_t memalloc_start, memalloc_end;       // start and end events of device memory allocation
        hipEvent_t memcpy_h2d_start, memcpy_h2d_end;   // start and end events of memory copy from host to device
        hipEvent_t kernel_start, kernel_end;           // start and end events of kernel execution   
        hipEvent_t memcpy_d2h_start, memcpy_d2h_end;   // start and end events of memory copy from device to host
        hipEvent_t memfree_start, memfree_end;         // start and end events of device memory free

        int vec_len = tg->init_states_vector.block_count;       // length (# of blocks) of state vector
        int total_input_bytes = 0;                              // sum of string length

        // Variables in host memory
        unsigned char *h_input;                         // total input string  
        int h_input_offset[array_size + 1];             // offsets of all input strings 
        ST_BLOCK *h_final_st_vec;                       // final active states of all strings

        // Variables in device memory
        unsigned char *d_input;                                         // total input string
        int *d_input_offset;                                            // offset of each input string
        Transition *d_transition_list;                                  // list of transition (source, destination) tuples
        //int *d_transition_offset;                                       // index of first transition trigger by each symbol   
        int *d_top_k_offset_per_symbol; 
        ST_BLOCK *d_init_st_vec, *d_final_st_vec, *d_lim_vec;     // state vectors

        // Create events
        if(profiler_mode){
                hipEventCreate(&memalloc_start);
                hipEventCreate(&memalloc_end);
                hipEventCreate(&memcpy_h2d_start);
                hipEventCreate(&memcpy_h2d_end);
                hipEventCreate(&kernel_start);
                hipEventCreate(&kernel_end);
                hipEventCreate(&memcpy_d2h_start);
                hipEventCreate(&memcpy_d2h_end);
                hipEventCreate(&memfree_start);
                hipEventCreate(&memfree_end);

                gettimeofday(&start_time, NULL);
        }

        for (int i = 0; i < array_size; i++) {
                h_input_offset[i] = total_input_bytes;
                total_input_bytes += input_bytes_array[i];
        }
        h_input_offset[array_size] = total_input_bytes;

        h_input = (unsigned char*)malloc(total_input_bytes);
        if (!h_input) {
                cerr << "Error: allocate host memory to store total input string" << endl;
                exit(-1);
        }

        // Copy each string into h_input to construct a big string
        for (int i = 0; i < array_size; i++) {
                memcpy(h_input + h_input_offset[i], h_input_array[i], input_bytes_array[i]);
        }

        // Allocate host memory
        h_final_st_vec = (ST_BLOCK*)malloc(sizeof(ST_BLOCK) * vec_len * array_size);
        if (!h_final_st_vec) {
                cerr << "Error: allocate host memory to store final state vectors" << endl;
                exit(-1);
        }
        
        // Allocate device memory
        if(profiler_mode) hipEventRecord(memalloc_start, 0);
        hipMalloc((void **)&d_input, total_input_bytes);
        hipMalloc((void **)&d_input_offset, sizeof(int) * (array_size + 1));
        hipMalloc((void **)&d_transition_list, sizeof(Transition) * tg->transition_count);
        //hipMalloc((void **)&d_transition_offset, sizeof(int) * (SYMBOL_COUNT + 1));
        hipMalloc((void **)&d_init_st_vec, sizeof(ST_BLOCK) * vec_len);
        hipMalloc((void **)&d_final_st_vec, sizeof(ST_BLOCK) * vec_len * array_size);
        hipMalloc((void **)&d_lim_vec, sizeof(ST_BLOCK)*vec_len * SYMBOL_COUNT * TOP_K);
        hipMalloc((void **)&d_top_k_offset_per_symbol, sizeof(int) * SYMBOL_COUNT * TOP_K);
        if(profiler_mode) hipEventRecord(memalloc_end, 0);
        
        // Copy input from host memory into device memory
        if(profiler_mode) hipEventRecord(memcpy_h2d_start, 0);
        hipMemcpy(d_input, h_input, total_input_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_input_offset, h_input_offset, sizeof(int) * (array_size + 1), hipMemcpyHostToDevice);
        hipMemcpy(d_transition_list, tg->transition_list, sizeof(Transition) * tg->transition_count, hipMemcpyHostToDevice);
        //hipMemcpy(d_transition_offset, tg->offset_per_symbol, sizeof(int) * (SYMBOL_COUNT + 1), hipMemcpyHostToDevice);
        if(hipSuccess!=hipMemcpyToSymbol(HIP_SYMBOL(c_transition_offset), tg->offset_per_symbol, sizeof(int) * (SYMBOL_COUNT + 1))) {cout<<"Error!\n"; exit(-1);}
        hipMemcpy(d_init_st_vec, tg->init_states_vector.vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
        hipMemcpy(d_top_k_offset_per_symbol, tg->top_k_offset_per_symbol, sizeof(int) * SYMBOL_COUNT * TOP_K, hipMemcpyHostToDevice);
        for (int i =0;i<SYMBOL_COUNT;i++){
                for(int j = 0; j<TOP_K; j++){
                        hipMemcpy(&d_lim_vec[i*TOP_K*vec_len+j*vec_len], tg->lim_vec[i][j].vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
                }
        }
        if(profiler_mode) hipEventRecord(memcpy_h2d_end, 0);

        // Calculate the size of shared memory (for 3 state vectors and transition offset)
        int shem = 3 * vec_len * sizeof(ST_BLOCK); // + sizeof(int) * (SYMBOL_COUNT + 1);

        // Launch kernel
        if(profiler_mode) hipEventRecord(kernel_start, 0);
        hipDeviceSetCacheConfig(hipFuncCachePreferShared);
        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
        TKO_kernel<<<array_size, threads_per_block, shem>>>(d_input,
                                                            d_input_offset,
                                                            d_transition_list,
                                                            //d_transition_offset,
                                                            d_init_st_vec,
                                                            d_final_st_vec,
                                                            d_top_k_offset_per_symbol,
                                                            d_lim_vec,
                                                            vec_len);
        if(profiler_mode) hipEventRecord(kernel_end, 0);
        if(profiler_mode) hipEventSynchronize(kernel_end);
        // Copy result from device memory into host memory
        if(profiler_mode) hipEventRecord(memcpy_d2h_start, 0);
        hipMemcpy(h_final_st_vec, d_final_st_vec, sizeof(ST_BLOCK) * vec_len * array_size, hipMemcpyDeviceToHost);
        if(profiler_mode) hipEventRecord(memcpy_d2h_end, 0);  

        // Get final active states and accept rules for each string
        vector<ST_T> final_states[array_size];
        vector<int> accept_rules[array_size];
        unordered_map<ST_T, vector<int> >::iterator itr;

        for (int i = 0; i < array_size; i++) {
                get_active_states(h_final_st_vec + i * vec_len, vec_len, final_states[i]);

                // Get all accept rules for string i
                for (int j = 0; j < final_states[i].size(); j++) {
                        // Get accept rules triggered by this state
                        itr = tg->accept_states_rules.find(final_states[i][j]);
                        if (itr != tg->accept_states_rules.end()) {
                                accept_rules[i].insert(accept_rules[i].end(), itr->second.begin(), itr->second.end());
                        }
                }                

                // Remove repeated accept rules for string i
                sort(accept_rules[i].begin(), accept_rules[i].end());
                accept_rules[i].erase(unique(accept_rules[i].begin(), accept_rules[i].end() ), accept_rules[i].end()); 
        }

        // Free device memory
        if(profiler_mode) hipEventRecord(memfree_start, 0);
        hipFree(d_input);
        hipFree(d_input_offset);
        hipFree(d_transition_list);
//        hipFree(d_transition_offset);
        hipFree(d_init_st_vec);
        hipFree(d_final_st_vec);
        hipFree(d_lim_vec);
        hipFree(d_top_k_offset_per_symbol);
        if(profiler_mode) hipEventRecord(memfree_end, 0);

        // Free host memory 
        free(h_final_st_vec);
        free(h_input);

        if(profiler_mode) gettimeofday(&end_time, NULL);

        if (show_match_result) show_results(array_size, final_states, accept_rules);

        Profiler(start_time, 
                end_time, 
                array_size, 
                memalloc_start, 
                memalloc_end,
                memcpy_h2d_start,
                memcpy_h2d_end,
                kernel_start,
                kernel_end,
                memcpy_d2h_start,
                memcpy_d2h_end,
                memfree_start,
                memfree_end);

        // Destroy events
        if(profiler_mode){
                hipEventDestroy(memalloc_start);
                hipEventDestroy(memalloc_end);
                hipEventDestroy(memcpy_h2d_start);
                hipEventDestroy(memcpy_h2d_end);
                hipEventDestroy(kernel_start);
                hipEventDestroy(kernel_end);
                hipEventDestroy(memcpy_d2h_start);
                hipEventDestroy(memcpy_d2h_end);
                hipEventDestroy(memfree_start);
                hipEventDestroy(memfree_end);
        }
        return accept_rules;
}
