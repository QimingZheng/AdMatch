#include <iostream>
#include <stdlib.h>
#include <string>
#include "string.h"

#include "include/ITA_regex.h"
#include "src/transition_graph.h"
#include "src/host_functions.h"

using namespace std;

void FLAG_VERIFICATION(ITA_FLAGS flag){
    int a,b,c;
    a = (flag&TKO_KERNEL)>0;
    b = (flag&AS_KERNEL)>0;
    c = (flag&INFA_KERNEL)>0;
    assert(a+b+c==1);
}

void allocScratch(struct ita_scratch &scratch){
    int vec_len = scratch.tg->init_states_vector.block_count;
    if(scratch.tg->kernel==iNFA){
        hipMalloc((void **)&(scratch.d_transition_list), sizeof(Transition) * scratch.tg->transition_count);
        hipMalloc((void **)&(scratch.d_transition_offset), sizeof(int) * (SYMBOL_COUNT + 1));
        hipMalloc((void **)&(scratch.d_init_st_vec), sizeof(ST_BLOCK) * vec_len);
        hipMalloc((void **)&(scratch.d_persis_st_vec), sizeof(ST_BLOCK) * vec_len); 

        hipMemcpy(scratch.d_transition_list, scratch.tg->transition_list, sizeof(Transition) * scratch.tg->transition_count, hipMemcpyHostToDevice);
        hipMemcpy(scratch.d_transition_offset, scratch.tg->offset_per_symbol, sizeof(int) * (SYMBOL_COUNT + 1), hipMemcpyHostToDevice);
        hipMemcpy(scratch.d_init_st_vec, scratch.tg->init_states_vector.vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
        hipMemcpy(scratch.d_persis_st_vec, scratch.tg->persis_states_vector.vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
    }
    if(scratch.tg->kernel==TKO_NFA){
        hipMalloc((void **)&(scratch.d_transition_list), sizeof(Transition) * scratch.tg->transition_count);
        hipMalloc((void **)&(scratch.d_init_st_vec), sizeof(ST_BLOCK) * vec_len);
        hipMalloc((void **)&(scratch.d_lim_vec), sizeof(ST_BLOCK)*vec_len * SYMBOL_COUNT * TOP_K);
        hipMalloc((void **)&(scratch.d_top_k_offset_per_symbol), sizeof(int) * SYMBOL_COUNT * TOP_K);

        hipMemcpy(scratch.d_transition_list, scratch.tg->transition_list, sizeof(Transition) * scratch.tg->transition_count, hipMemcpyHostToDevice);
        hipMemcpy(scratch.d_init_st_vec, scratch.tg->init_states_vector.vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
        hipMemcpy(scratch.d_top_k_offset_per_symbol, scratch.tg->top_k_offset_per_symbol, sizeof(int) * SYMBOL_COUNT * TOP_K, hipMemcpyHostToDevice);
        for (int i =0;i<SYMBOL_COUNT;i++){
            for(int j = 0; j<TOP_K; j++){
                    hipMemcpy(&(scratch.d_lim_vec[i*TOP_K*vec_len+j*vec_len]), scratch.tg->lim_vec[i][j].vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
            }
        }
    }
    if(scratch.tg->kernel==AS_NFA){
        hipMalloc((void **)&(scratch.d_init_st_vec), sizeof(ST_BLOCK) * vec_len);
        hipMalloc((void **)&(scratch.d_transition_table), sizeof(ST_BLOCK) * vec_len * scratch.tg->state_count * SYMBOL_COUNT);
        hipMalloc((void **)&(scratch.d_transition_list), sizeof(Transition) * scratch.tg->wb_transition_count);

        hipMemcpy(scratch.d_init_st_vec, scratch.tg->init_states_vector.vector, sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
        hipMemcpy(scratch.d_transition_list, scratch.tg->transition_list, sizeof(Transition) * scratch.tg->wb_transition_count, hipMemcpyHostToDevice);
        for(int i=0;i<SYMBOL_COUNT;i++)
        {
        for(int j=0;j<scratch.tg->state_count;j++)
        {
                hipMemcpy(&(scratch.d_transition_table[vec_len*(i*scratch.tg->state_count+j)]),
                        scratch.tg->transition_table[i*scratch.tg->state_count+j].vector,
                        sizeof(ST_BLOCK) * vec_len, hipMemcpyHostToDevice);
        }
        }
    }
}

void freeScratch(struct ita_scratch &scratch){
    if(scratch.tg->kernel==iNFA){
        hipFree(scratch.d_transition_list);
        hipFree(scratch.d_transition_offset);
        hipFree(scratch.d_init_st_vec);
        hipFree(scratch.d_persis_st_vec);
    }
    if(scratch.tg->kernel==TKO_NFA){
        hipFree(scratch.d_transition_list);
        hipFree(scratch.d_lim_vec);
        hipFree(scratch.d_init_st_vec);
        hipFree(scratch.d_top_k_offset_per_symbol);
    }
    if(scratch.tg->kernel==AS_NFA){
        hipFree(scratch.d_transition_list);
        hipFree(scratch.d_init_st_vec);
        hipFree(scratch.d_transition_table);
    }
}


void Scan(ITA_FLAGS flag, char *nfa, char *text, vector<int> *accepted_rules){
    FLAG_VERIFICATION(flag);
    Kernel_Type kernel;
    if (flag&INFA_KERNEL) kernel=iNFA;
    if (flag&AS_KERNEL) kernel=AS_NFA;
    if (flag&TKO_KERNEL) kernel=TKO_NFA;

    TransitionGraph tg(kernel);

    if (!tg.load_nfa_file(nfa)) {
        cerr << "Error: load NFA file " << nfa << endl;
        exit(-1);
    }

    unsigned char *h_input_array[1];
    int input_bytes_array[1];
    h_input_array[0]=(unsigned char*)text;
    input_bytes_array[0]=strlen(text);

    run_nfa(&tg, h_input_array, input_bytes_array, 1, 1024, flag&SHOW_RESULTS, flag&PROFILER_MODE, accepted_rules);
}

void BatchedScan(ITA_FLAGS flag, char *nfa, char **text, int *text_len, int str_count, vector<int> *accepted_rules){
    FLAG_VERIFICATION(flag);
    Kernel_Type kernel;
    if (flag&INFA_KERNEL) kernel=iNFA;
    if (flag&AS_KERNEL) kernel=AS_NFA;
    if (flag&TKO_KERNEL) kernel=TKO_NFA;

    TransitionGraph tg(kernel);

    if (!tg.load_nfa_file(nfa)) {
        cerr << "Error: load NFA file " << nfa << endl;
        exit(-1);
    }

    unsigned char *h_input_array[str_count];
    
    for (int i = 0; i < str_count; i++) {
        h_input_array[i] = (unsigned char *) text[i];
    }

    run_nfa(&tg, h_input_array, text_len, str_count, 32, flag&SHOW_RESULTS, flag&PROFILER_MODE, accepted_rules);

}
