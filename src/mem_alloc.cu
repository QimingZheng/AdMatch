#include "src/mem_alloc.h"

// Allocate host memory. Return true if the allocation succeeds.
bool alloc_host(void **ptr, size_t size) {
    return hipHostMalloc(ptr, size) == hipSuccess;
}

// Free host memory
void free_host(void *ptr) { hipHostFree(ptr); }
